
#include <hip/hip_runtime.h>
template <typename T>  __global__  void kernelgpuSource(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T xdg1 = xdg[0*ng+i];
		T xdg2 = xdg[1*ng+i];
		f[0*ng+i] = sin(xdg1*3.141592653589793)*sin(xdg2*3.141592653589793)*1.973920880217872E1;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuSource(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuSource<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuSource(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int);
template void gpuSource(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int);
